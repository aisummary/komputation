#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__device__ float tanh (float x) {
    return (2.0 / (1.0 + expf(-2.0*x))) - 1.0;
}

__global__ void tanhKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float *source,
    float *destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = tanh(source[indexEntry]);
            }
        }
        else {
            setToNan(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}