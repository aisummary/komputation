#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__global__ void backwardDropoutKernel (
    int batchSize,
    int numberEntriesPerInstance,
    int numberRows,
    int numberIterations,
    float* chain,
    float* mask,
    float* result) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                result[indexEntry] = chain[indexEntry] * mask[indexEntry];
            }
        }
        else {
            setToNan(result, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}