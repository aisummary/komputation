#include "hip/hip_runtime.h"
__global__ void momentumKernel (
    int numberIterations,
    int* parameterIndices,
    int* counts,
    int parameterSize,
    float* parameters,
    float* gradient,
    float learningRate,
    float momentum,
    float* history,) {

    int startEntry = (blockIdx.y * blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    if(startEntry < parameterSize) {

        int gradientIndex = blockIdx.x;
        int parameterIndex = parameterIndices[gradientIndex];

        if(parameterIndex != -1) {

            int startParameter = parameterIndex * parameterSize + startEntry;
            int startGradient = gradientIndex * parameterSize + startEntry;

            float scalingFactor = 1.0 / (float)counts[gradientIndex];

            for(int indexParameter = startParameter, indexGradient = startGradient; indexParameter < startParameter + numberIterations; indexParameter++, indexGradient++) {

                float update = momentum * history[indexParameter] - scalingFactor * learningRate * gradient[indexGradient];

                history[indexParameter] = update;
                parameters[indexParameter] += update;

            }

        }

    }

}