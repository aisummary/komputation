#include "hip/hip_runtime.h"
__global__ void stochasticGradientDescentKernel (
    int numberIterations,
    int* parameterIndices,
    int* counts,
    int parameterSize,
    float* parameters,
    float* gradient,
    float learningRate,) {

    int startEntry = (blockIdx.y * blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    if(startEntry < parameterSize) {

        int gradientIndex = blockIdx.x;
        int parameterIndex = parameterIndices[gradientIndex];

        if(parameterIndex != -1) {

            int startParameter = parameterIndex * parameterSize + startEntry;
            int startGradient = gradientIndex * parameterSize + startEntry;

            float scalingFactor = 1.0 / (float)counts[gradientIndex];

            for(int indexParameter = startParameter, indexGradient = startGradient; indexParameter < startParameter + numberIterations; indexParameter++, indexGradient++) {

                parameters[indexParameter] -= scalingFactor * learningRate * gradient[indexGradient];

            }

        }

    }

}