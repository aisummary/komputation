#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__global__ void dropoutRuntimeKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float keepProbability,
    float* input,
    float* result) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                result[indexEntry] = keepProbability * input[indexEntry];
            }
        }
        else {
            setToNaN(result, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}