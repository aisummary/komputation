#include "hip/hip_runtime.h"
#include "../../hip/hip_runtime.h"
#include "../../symbols/NaN.cuh"

__inline__ __device__  int xorShift(int seed) {
    int updated = seed;

    updated ^= updated << 13;
    updated ^= updated >> 17;
    updated ^= updated << 5;

    return updated;
}

/*
    dropout probability is 1 - keep probability and should be less than 1.
    Adding 2147483648 ensures that the result is non-negative (from 0 to 2 * upper integer bound).
    Division by 4294967295.0 returns a percentage (from 0 to 1).
    Subtraction by the probability dropout probability returns either a positive or a negative number.
    Drop out if the number is negative.
*/
__device__ float generateMask(float seed, float dropoutProbability) {

    return ceilf((seed + 2147483648.0f) / 4294967295.0f - dropoutProbability);

}

__global__ void dropoutTrainingKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float dropoutProbability,
    float* input,
    int* seeds,
    float* masks,
    float* result) {
    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                int newSeed = xorShift(seeds[indexEntry]);
                seeds[indexEntry] = newSeed;

                float mask = generateMask((float)newSeed, dropoutProbability);
                masks[indexEntry] = mask;

                result[indexEntry] = mask * input[indexEntry];
            }
        }
        else {
            setToNaN(result, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}