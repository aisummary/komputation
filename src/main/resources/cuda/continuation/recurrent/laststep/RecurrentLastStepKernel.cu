#include "hip/hip_runtime.h"
#include "continuation/recurrent/Recurrent.cuh"
#include "symbols/NaN.cuh"

__global__ void recurrentLastStepKernel (
    int activationFunction,
    int maximumEntriesPerInstance,
    int hiddenDimension,
    int numberIterations,
    float* projectedInput,
    float* preActivation,
    float* previousStateWeights,
    int* lengths,
    int maximumLength,
    float* hiddenStates,
    float* result) {

    int instanceIndex = blockIdx.x;

    int firstInstanceEntryIndex = instanceIndex * maximumEntriesPerInstance;

    int startEntryIndex = threadIdx.x * numberIterations;
    // Do not go past the hidden dimension
    int exclusiveEndEntryIndex = min(startEntryIndex + numberIterations, hiddenDimension);

    extern __shared__ float sharedData[];

    forwardFirstStep(projectedInput, preActivation, hiddenStates, firstInstanceEntryIndex, sharedData, startEntryIndex, exclusiveEndEntryIndex, activationFunction);

    __syncthreads();

    int length = lengths[instanceIndex];

    int firstStateEntryIndex = firstInstanceEntryIndex;
    for(int step = 1; step < length; step++) {
        firstStateEntryIndex += hiddenDimension;

        forwardOtherStep(projectedInput, preActivation, hiddenStates, sharedData, previousStateWeights, firstStateEntryIndex, startEntryIndex, exclusiveEndEntryIndex, activationFunction, hiddenDimension);

        __syncthreads();
    }

    copyCooperatively(sharedData, 0, result, instanceIndex * hiddenDimension, startEntryIndex, exclusiveEndEntryIndex);

    for(int step = length; step < maximumLength; step++) {
        firstStateEntryIndex += hiddenDimension;

        setToNaN(result, firstStateEntryIndex + startEntryIndex, firstStateEntryIndex + exclusiveEndEntryIndex);
    }

}