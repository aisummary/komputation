#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__inline__ __device__ float relu (float x)
{
    return fmaxf(x, 0.0);
}

__global__ void reluKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* source,
    float* destination) {
    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = relu(source[indexEntry]);
            }
        }
        else {
            setToNan(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }
}