#include "hip/hip_runtime.h"
#include "../../hip/hip_runtime.h"
#include "../../symbols/NaN.cuh"
#include "Tanh.cuh"

__global__ void backwardTanhKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* forward,
    float* chain,
    float* destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = backwardTanh(forward[indexEntry], chain[indexEntry]);
            }
        }
        else {
            setToNaN(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}