#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__inline__ __device__ float backwardTanh (float forward, float chain) {
    return chain * (1.0 - powf(forward, 2.0));
}

__global__ void backwardTanhKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* forward,
    float* chain,
    float* destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = backwardTanh(forward[indexEntry], chain[indexEntry]);
            }
        }
        else {
            setToNaN(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}