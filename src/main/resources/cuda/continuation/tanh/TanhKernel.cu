#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"
#include "continuation/tanh/Tanh.cuh"

__global__ void tanhKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* source,
    float* destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = tanh(source[indexEntry]);
            }
        }
        else {
            setToNaN(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}