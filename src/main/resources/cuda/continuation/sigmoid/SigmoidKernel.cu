#include "hip/hip_runtime.h"
#include "../../hip/hip_runtime.h"
#include "../../symbols/NaN.cuh"
#include "Sigmoid.cuh"

__global__ void sigmoidKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* source,
    float* destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = sigmoid(source[indexEntry]);
            }
        }
        else {
            setToNaN(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}