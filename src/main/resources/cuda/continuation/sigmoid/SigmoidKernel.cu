#include "hip/hip_runtime.h"
#include "symbols/NaN.cuh"

__inline__ __device__ float sigmoid (float x) {
    return 1.0 / (1.0 + expf (-x));
}

__global__ void sigmoidKernel (
    int batchSize,
    int numberRows,
    int numberEntriesPerInstance,
    int numberIterations,
    float* source,
    float* destination) {

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int startRowWithinColumn = threadIdx.x * numberIterations;

    int firstEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + startRowWithinColumn;
    int startNextColumn = startInstanceWithinBatch + startColumnWithinInstance + numberRows;

    if(firstEntryWithinBatch < startNextColumn) {
        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextColumn);

        if(indexInstance < batchSize) {
            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {
                destination[indexEntry] = sigmoid(source[indexEntry]);
            }
        }
        else {
            setToNan(destination, firstEntryWithinBatch, lastEntryWithinBatch);
        }
    }

}