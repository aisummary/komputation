
#include <hip/hip_runtime.h>
__global__ void replaceNaNKernel(
    int numberEntries,
    int numberEntriesPerInstance,
    int numberIterations,
    float* source,
    float* destination) {

    int instanceIndex = blockIdx.x;

    int instanceStart = instanceIndex * numberEntriesPerInstance;
    int startWithinThread = instanceStart + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    for(int index = startWithinThread; index < min(startWithinThread + numberIterations, instanceStart + numberEntriesPerInstance); index++) {
        float currentValue = source[index];
        destination[index] = isnan(currentValue) ? 0.0 : currentValue;
    }
}