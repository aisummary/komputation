#include "hip/hip_runtime.h"
#include "../../hip/hip_runtime.h"
#include "../../symbols/NaN.cuh"

__global__ void backwardSquaredLossKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float* predictions, float* targets, float* result) {

    // What's the first entry index within the instance that this thread should operate on?
    int startIndexWithinInstance = blockIdx.y * (blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    // Continue if this index is smaller than the dimension of the instance.
    if(startIndexWithinInstance < numberEntriesPerInstance) {
        // What's the first entry index within the batch that this thread should operate on?
        int startIndexWithinBatch = blockIdx.x * numberEntriesPerInstance + startIndexWithinInstance;

        // Is the instance greater than the current batch size?
        if(blockIdx.x >= batchSize) {
            setToNaN(result, startIndexWithinBatch, numberIterations);
        }
        else {
            for(int indexEntry = startIndexWithinBatch; indexEntry < startIndexWithinBatch + numberIterations; indexEntry++) {
                result[indexEntry] = predictions[indexEntry] - targets[indexEntry];
            }
        }
    }

}