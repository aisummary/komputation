#include "hip/hip_runtime.h"
#include "reduction/SumReduction.cuh"

/*
    Example:

                  0.2 0.7
    predictions = 0.3 0.1
                  0.5 0.2

    targets = 0.0 1.0
              1.0 0.0
              0.0 0.0

    number of threads per column = the smallest number equal to the number of rows/categories or greater than the number of rows/categories that is a power of 2 = 4
    number of blocks = number of columns/steps

    shared data in the first block:
    [ 0 * 0.2, 1 * 0.3, 0 * 0.5, 0 * 0.0 ]
    = [ 0, 0.3, 0.0, 0.0 ]

    shared data in the second block:
    [ 1.0 * 0.7, 0.0 * 0.1, 0.0 * 0.2, 0 * 0.0 ]
    = [ 0.7, 0.0, 0.0, 0.0 ]

    parallel sum reduction in the first block:
    [ 0.3, 0.3, 0.0, 0.0 ]

    parallel sum reduction in the second block:
    [ 0.7, 0.0, 0.0, 0.0 ]

    parallel product reduction of the sums of each block:
    0.3 * 0.7 = 0.21

    Negative log:
    -log(0.21) = 0.677780705

*/

__global__ void crossEntropyLossKernel (int batchSize, int numberRows, int numberEntriesPerInstance, int numberIterations, float* predictions, float* targets, float* result)
{

    int startIndexWithinColumn = threadIdx.x * numberIterations;

    extern __shared__ float sharedData[];

    int indexInstance = blockIdx.x;
    int indexColumn = blockIdx.y;

    int startIndexWithinInstance = indexColumn * numberRows + startIndexWithinColumn;
    int startIndexWithinBatch = indexInstance * numberEntriesPerInstance + startIndexWithinInstance;

    int indexColumnInBatch = indexInstance * gridDim.y + indexColumn;

    if(indexInstance < batchSize) {
        float thisValue = 0.0;

        if(startIndexWithinColumn < numberRows) {
            thisValue = targets[startIndexWithinBatch] * predictions[startIndexWithinBatch];

            if(numberIterations > 1) {
                for(int indexEntry = startIndexWithinBatch + 1; indexEntry < startIndexWithinBatch + numberIterations; indexEntry++) {
                    thisValue += targets[indexEntry] * predictions[indexEntry];
                }
            }
        }

        int warpId = threadIdx.x / warpSize;
        int laneId = threadIdx.x % warpSize;

        reduceToSum(thisValue, warpId, laneId, sharedData);

        if(threadIdx.x == 0) {
            result[indexColumnInBatch] = -logf(sharedData[0]);
        }
    }
    else {
        if(threadIdx.x == 0) {
            result[indexColumnInBatch] = 0.0;
        }
    }

}